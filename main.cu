#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>
#include <memory>
#include <stdint.h>

template <class T> 
void restore(T* reciever, T* sender, int Size){
    hipMemcpy(reciever, sender, Size * sizeof(T), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

template <class T>
void send(T* reciever, T* sender, int Size){
    hipMemcpy(reciever, sender, Size * sizeof(T), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

__global__ void kernelSum(const uint8_t* deviceInput, uint32_t* deviceOutput, uint32_t H, uint32_t W, uint32_t BatchLength){
    uint32_t sx = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t tx = threadIdx.x;
    uint32_t stride;
    printf("%u %u\n", sx, tx);
    __shared__ uint32_t sMemory[64];

    sMemory[tx] = deviceInput[sx];

    __syncthreads();
    for(stride = blockDim.x / 2; stride >= 1; stride >>=1){
        if (tx < stride) {
            sMemory[tx] = sMemory[tx] + sMemory[tx + stride];
        }
        __syncthreads();
    }
    
    if (tx==0U) { 
        deviceOutput[blockIdx.x] = sMemory[tx];
        printf("sum : %d\n", deviceOutput[blockIdx.x]);
    }
}

std::vector<uint8_t> GenerateArry(uint64_t H, uint64_t W, uint64_t BatchLength){
    uint64_t size = H * W * BatchLength;
    std::vector<uint8_t> arr(size, 1);
    return arr;
};

void BatchSum(std::vector<uint8_t> Arr1d, uint64_t H, uint64_t W, uint64_t BatchLength){
    // init
    uint8_t* deviceInput;
    uint32_t* deviceOutput;
    uint32_t* results;
    dim3 block(64, 1, 1);
    dim3 grid(1, 1, 1);

    results = (uint32_t*)malloc(BatchLength * sizeof(uint32_t));
    hipMalloc((void**)&deviceInput, H * W * BatchLength * sizeof(uint8_t));
    hipMalloc((void**)&deviceOutput, H * W * BatchLength * sizeof(uint32_t));
    hipDeviceSynchronize();
    
    send(deviceInput, Arr1d.data(), H * W * BatchLength);
    kernelSum<<<grid, block>>>(deviceInput, deviceOutput, H, W, BatchLength);
    restore(results, deviceOutput, H * W * BatchLength);

    free(results);
    hipFree(deviceInput);
    hipFree(deviceOutput);
}

int main(){
    uint64_t H = 8;
    uint64_t W = 8;
    uint64_t BatchLength = static_cast<uint64_t>(1);
    //printf("%llu %llu %llu %llu\n", H, W, BatchLength, H * W * BatchLength);
    std::vector<uint8_t> Arr1d = GenerateArry(H, W, BatchLength);
    BatchSum(Arr1d, H, W, BatchLength);
    printf("End\n");
};