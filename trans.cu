#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <windows.h>
using namespace std;


template<class T>
__global__ void transpose(T* device_ouput, const T* device_input, int w, int h){
    const int sx = blockDim.x * blockIdx.x + threadIdx.x;
    const int sy = blockDim.y * blockIdx.y + threadIdx.y;
    if(w<=sx||h<=sy){return;}
    device_ouput[sx*h+sy] = device_input[sy*w+sx];
}

template<class T>
void send(T *reciver, T *sender, int data_size){
    hipMemcpy(reciver, sender, sizeof(T)*data_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

template<class T>
void restore(T *reciver, T *sender, int data_size){
    hipMemcpy(reciver, sender, sizeof(T)*data_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}



int main(){
    int H=4;
    int W=4;
    dim3 block(1, 1, 1);
    dim3 grid(H, W, 1);

    // original_data and result; 
    int *A = (int*)malloc(H*W*sizeof(int));
    int *At = (int*)malloc(H*W*sizeof(int));
    
    int *device_input;
    int *device_output;
    hipMalloc((void**)&device_input, H*W*sizeof(int));
    hipMalloc((void**)&device_output, H*W*sizeof(int));
    // init
    for(int i=0;i<H*W;i++){A[i]=i;}
    // task
    send(device_input, A, H*W);
    transpose<<<block,grid>>>(device_output, device_input, W, H);
    restore(At, device_output, H*W);

    for(int i=0;i<H;i++){
        for(int j=0;j<W;j++){
            cout<<At[j+W*i]<<" ";
        }
        cout<<endl;
    }
    free(A);
    free(At);
    hipFree(device_output);
    hipFree(device_input);
}